#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <fcntl.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

#include <fstream>
#include <iostream>

#include "cufile.h"

#define KB(x) ((x)*1024L)
#define TESTFILE "/mnt/test"

__global__ void hello(char *str) {
	printf("Hello World!\n");
	printf("buf: %s\n", str);
}

__global__ void strrev(char *str, int *len) {
	int size = 0;
	while (str[size] != '\0') {
		size++;
	}
	for(int i=0;i<size/2;++i) {
		char t = str[i];
		str[i] = str[size-1-i];
		str[size-1-i] = t;
	}
	/*
	printf("buf: %s\n", str);
	printf("size: %d\n", size);
	*/
	*len = size;
}

__global__ void g_reverse_int(unsigned int i) {
  unsigned char ch1, ch2, ch3, ch4;
  ch1 = i & 255;
  ch2 = (i >> 8) & 255;
  ch3 = (i >> 16) & 255;
  ch4 = (i >> 24) & 255;
//   return ((unsigned int)ch1 << 24) + ((unsigned int)ch2 << 16) +
//          ((unsigned int)ch3 << 8) + ch4;
}

unsigned int reverse_int(unsigned int i) {
  unsigned char ch1, ch2, ch3, ch4;
  ch1 = i & 255;
  ch2 = (i >> 8) & 255;
  ch3 = (i >> 16) & 255;
  ch4 = (i >> 24) & 255;
  return ((unsigned int)ch1 << 24) + ((unsigned int)ch2 << 16) +
         ((unsigned int)ch3 << 8) + ch4;
}

void test_stream(char* file_name) {
    int fd;
    int ret;
    
    fd = open(file_name, O_RDWR | O_DIRECT);                                

    if (fd != -1) {
        int *sys_len;
        int *gpu_len;
        char *system_buf;
        char *gpumem_buf;

        unsigned int magic_number = 0;
        unsigned int number_of_images = 0;
        unsigned int n_rows = 0;
        unsigned int n_cols = 0;


        read(fd, (char*)&magic_number, sizeof(magic_number));
        read(fd, (char*)&number_of_images, sizeof(number_of_images));
        read(fd, (char*)&n_rows, sizeof(n_rows));
        read(fd, (char*)&n_cols, sizeof(n_cols));
        magic_number = reverse_int(magic_number);
        number_of_images = reverse_int(number_of_images);
        n_rows = reverse_int(n_rows);
        n_cols = reverse_int(n_cols);

        std::cout << file_name << std::endl;
        std::cout << "magic number = " << magic_number << std::endl;
        std::cout << "number of images = " << number_of_images << std::endl;
        std::cout << "rows = " << n_rows << std::endl;
        std::cout << "cols = " << n_cols << std::endl;

        // system_buf = (char*)malloc(bufsize);
        // sys_len = (int*)malloc(KB(1));
		int bufsize = n_rows * n_cols * sizeof(char);
		int n_bufsize = n_rows * n_cols * sizeof(float);

        hipMalloc(&gpumem_buf, bufsize);
        hipMalloc(&gpu_len, KB(1));
		off_t file_offset = 0;
		off_t mem_offset = 0;
        CUfileDescr_t cf_desc; 
        CUfileHandle_t cf_handle;

        cuFileDriverOpen();

        cf_desc.handle.fd = fd;
        cf_desc.type = CU_FILE_HANDLE_TYPE_OPAQUE_FD;
		cuFileHandleRegister(&cf_handle, &cf_desc);
		cuFileBufRegister((char*)gpumem_buf, bufsize, 0);



        // std::vector<unsigned char> image(n_rows * n_cols);
        // std::vector<float> normalized_image(n_rows * n_cols);

        for (int i = 0; i < number_of_images; i++) {
            // file.read((char*)&image[0], sizeof(unsigned char) * n_rows * n_cols);

			ret = cuFileRead(cf_handle, (char*)gpumem_buf, bufsize, file_offset, mem_offset);
			file_offset += bufsize;
			mem_offset += bufsize;

            // for (int i = 0; i < n_rows * n_cols; i++) {
            // normalized_image[i] = (float)image[i] / 255 - 0.5;
            // }
            // output.push_back(normalized_image);
        }

		hipFree(gpumem_buf);
		hipFree(gpu_len);		
		close(fd);
		cuFileDriverClose();
    }
}

void test(char * file_name) {
	int fd;
	int ret;
	int *sys_len;
	int *gpu_len;
	char *system_buf;
	char *gpumem_buf;

	int bufsize=KB(8);
	int parasize=KB(1);

	system_buf = (char*)malloc(bufsize);
	sys_len = (int*)malloc(parasize);

	hipMalloc(&gpumem_buf, bufsize);
	hipMalloc(&gpu_len, parasize);
	off_t file_offset = 0;
	off_t mem_offset = 0;

	CUfileDescr_t cf_desc; 
	CUfileHandle_t cf_handle;

	cuFileDriverOpen();
	fd = open(file_name, O_RDWR | O_DIRECT);

	cf_desc.handle.fd = fd;
	cf_desc.type = CU_FILE_HANDLE_TYPE_OPAQUE_FD;

	cuFileHandleRegister(&cf_handle, &cf_desc);
	cuFileBufRegister((char*)gpumem_buf, bufsize, 0);

	ret = cuFileRead(cf_handle, (char*)gpumem_buf, bufsize, file_offset, mem_offset);
	if (ret < 0) {
		printf("cuFileRead failed : %d", ret); 
	}

	/*
	hello<<<1,1>>>(gpumem_buf);
	*/
	strrev<<<1,1>>>(gpumem_buf, gpu_len);

	hipMemcpy(sys_len, gpu_len, parasize, hipMemcpyDeviceToHost);
	printf("sys_len : %d\n", *sys_len); 
	ret = cuFileWrite(cf_handle, (char*)gpumem_buf, *sys_len, file_offset, mem_offset);
	if (ret < 0) {
		printf("cuFileWrite failed : %d", ret); 
	}

	hipMemcpy(system_buf, gpumem_buf, bufsize, hipMemcpyDeviceToHost);
	printf("system_buf: %s\n", system_buf);
	printf("See also %s\n", file_name);

	cuFileBufDeregister((char*)gpumem_buf);

	hipFree(gpumem_buf);
	hipFree(gpu_len);
	free(system_buf);
	free(sys_len);

	close(fd);
	cuFileDriverClose();
}

int main(int argc, char *argv[])
{
	test(argv[1]);
	// char * mnist_data="/home/steven/dev/DataLoaders_DALI/cuda-neural-network/build/mnist_data/train-images-idx3-ubyte";
	// test_stream(mnist_data);
}
